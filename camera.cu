#include "hip/hip_runtime.h"
#ifndef CAMERA_H
#define CAMERA_H

#include <iostream>
#include <random>
#include "glm/vec3.hpp"
#include "glm/gtc/matrix_transform.hpp"
#include "hittable.cuh"
using namespace glm;

float ASPECT_RATIO = 16.0/9.0;

const int IMG_WIDTH = 1280;
const int IMG_HEIGHT = IMG_WIDTH/ASPECT_RATIO;

const float V_WIDTH = 2.0;
const float V_HEIGHT = V_WIDTH/((float)IMG_WIDTH/IMG_HEIGHT);

const int PIXEL_SAMPLES = 8;
const float PIXEL_SCALE = 1.0/PIXEL_SAMPLES;

class Camera {
public:
    __host__ Camera() {}
    __host__ Camera(vec3 c_p, float f_l): camera_pos(c_p), focal_length(f_l) {
        initializeViewport();
    }

    vec3 camera_pos;
    float focal_length;

    vec3 viewport_top_left;
    vec3 viewport_u;
    vec3 viewport_v;
    vec3 pixels_top_left;

    __host__ void initializeViewport() {
        viewport_top_left = camera_pos + vec3(0,0,focal_length) + vec3(-V_WIDTH/2.0, V_HEIGHT/2.0, 0);
        viewport_u = vec3(V_WIDTH/IMG_WIDTH, 0, 0);
        viewport_v = vec3(0, -V_HEIGHT/IMG_HEIGHT, 0);
        pixels_top_left = viewport_top_left + viewport_u*0.5f + viewport_v*0.5f;
    }
};

__constant__ Sphere d_sphere;
__constant__ Camera d_cam;

__device__ vec3 rayMarch(vec3 current_pos, const vec3& ray_direction) {
    float dist = d_sphere.SDF(current_pos);
    for (int k = 0; k < 50; k++) {
        current_pos = current_pos + ray_direction*dist;
        dist = d_sphere.SDF(current_pos);
        if (dist < 0.001) {
            return vec3(255,0,0);
        }
    }
    return vec3(0,0,0);
}

__global__ void testRender(vec3* d_pixels, int HEIGHT, int WIDTH) {
    int i = blockDim.y*blockIdx.y + threadIdx.y;
    int j = blockDim.x*blockIdx.x + threadIdx.x;
    if (i < HEIGHT && j < WIDTH) {
        vec3 pixel = d_cam.pixels_top_left + d_cam.viewport_u*(float)j + d_cam.viewport_v*(float)i;
        vec3 ray_direction = (pixel-d_cam.camera_pos)/distance(pixel, d_cam.camera_pos);
        d_pixels[i*WIDTH + j] = rayMarch(d_cam.camera_pos, ray_direction);
    } 
}

__host__ vec3* loadScene(const Camera& h_cam, const Sphere& h_sphere) {
    hipMemcpyToSymbol(HIP_SYMBOL(d_sphere), &h_sphere, sizeof(Sphere));
    hipMemcpyToSymbol(HIP_SYMBOL(d_cam), &h_cam, sizeof(Camera));
    
    vec3* h_pixels = new vec3[IMG_HEIGHT*IMG_WIDTH];
    vec3* d_pixels;
    hipMalloc((void**)&d_pixels, IMG_HEIGHT*IMG_WIDTH*sizeof(vec3));
    hipMemcpy(d_pixels, h_pixels, IMG_HEIGHT*IMG_WIDTH*sizeof(vec3), hipMemcpyHostToDevice);

    dim3 blocksPerGrid((IMG_WIDTH+8-1)/8, (IMG_HEIGHT+8-1)/8);
    dim3 threadsPerBlock(8,8);
    testRender<<<blocksPerGrid, threadsPerBlock>>>(d_pixels, IMG_HEIGHT, IMG_WIDTH);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    std::cerr << hipGetErrorName(err) << ": " << hipGetErrorString(err) << '\n';
    hipMemcpy(h_pixels, d_pixels, IMG_HEIGHT*IMG_WIDTH*sizeof(vec3), hipMemcpyDeviceToHost);
    hipFree(d_pixels);
    std::clog << "Rendering Complete." << '\n';
    return h_pixels;
}

__host__ void outputImage(vec3* pixels) {
    std::cout << "P3\n" << IMG_WIDTH << ' ' << IMG_HEIGHT << "\n255\n";
    for (int i = 0; i < IMG_HEIGHT; i++) {
        for (int j = 0; j < IMG_WIDTH; j++) {
            std::cout << pixels[i*IMG_WIDTH + j].x << ' ' << pixels[i*IMG_WIDTH + j].y << ' ' << pixels[i*IMG_WIDTH + j].z << "\n";
        }
    }
    std::clog << "Image Creation Complete." << '\n';
}

#endif